
#include <hip/hip_runtime.h>
// na napravi izračunamo vsote kvadratov za vsak blok:
//		uporabimo skupni pomnilnik, dinamična rezervacija
//		redukcija po drevesu, korak se zmanjšuje, v snopu ne potrebujemo sinhronizacije

#ifdef __cplusplus
extern "C" {
#endif

__global__ void vectorDistanceLD4(float *p, const float *a, const float *b, int len) {
	// skupni pomnilnik niti v bloku
	extern __shared__ float part[];
	part[threadIdx.x] = 0.0;

	// kvadriranje razlike
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	float diff;
	while (gid < len) {
		diff = a[gid] - b[gid];
		part[threadIdx.x] += diff * diff;
		gid += gridDim.x * blockDim.x;
	}

	// počakamo, da vse niti zaključijo
	__syncthreads();

	// izračunamo delno vsoto za blok niti
	int idxStep;
	for(idxStep = blockDim.x >> 1; idxStep > 32 ; idxStep >>= 1) {
		if (threadIdx.x < idxStep)
			part[threadIdx.x] += part[threadIdx.x+idxStep];
		__syncthreads();
	}
	for( ; idxStep > 0 ; idxStep >>= 1 ) {
		if (threadIdx.x < idxStep)
			part[threadIdx.x] += part[threadIdx.x+idxStep];
		__syncwarp();
	}

	if (threadIdx.x == 0)
		p[blockIdx.x] = part[0];
}

#ifdef __cplusplus
}
#endif
