
#include <hip/hip_runtime.h>
// računanje razlike elementov dveh vektorjev
// dobra rešitev, neodvisna od števila blokov 

#ifdef __cplusplus
extern "C" {
#endif

__global__ void vectorSubtract4(float *c, const float *a, const float *b, int len) {
	// določimo globalni indeks elementov
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	// če je niti manj kot je dolžina vektorjev, morajo nekatere izračunati več razlik
	while (gid < len) {
		c[gid] = a[gid] - b[gid];
		gid += gridDim.x * blockDim.x;
	}
}

#ifdef __cplusplus
}
#endif
