
#include <hip/hip_runtime.h>
// bitonično urejanje

#ifdef __cplusplus
extern "C" {
#endif

__device__ void bitonicSort(int *a, int len, int k, int j) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;    
    while (gid < len/2) {
		int i1 = 2*j * (int)(gid / j) + (gid % j);	// prvi element
		int i2 = i1 ^ j;							// drugi element
		int dec = i1 & k;							// smer urejanja (padajoče: dec != 0)
		if ((dec == 0 && a[i1] > a[i2]) || (dec != 0 && a[i1] < a[i2])) {
			int temp = a[i1];
			a[i1] = a[i2];
			a[i2] = temp;
		}
		gid += gridDim.x * blockDim.x;
	}
}

__global__ void bitonicSortStart(int *a, int len) {
	for (int k = 2; k <= 2 * blockDim.x; k <<= 1) 
		for (int j = k/2; j > 0; j >>= 1) {
			bitonicSort(a, len, k, j);
			__syncthreads();
	}
}

__global__ void bitonicSortMiddle(int *a, int len, int k, int j) {
	bitonicSort(a, len, k, j);
}

__global__ void bitonicSortFinish(int *a, int len, int k) {
	for (int j = blockDim.x; j > 0; j >>= 1) {
		bitonicSort(a, len, k, j);
		__syncthreads();
	}
}


#ifdef __cplusplus
}
#endif