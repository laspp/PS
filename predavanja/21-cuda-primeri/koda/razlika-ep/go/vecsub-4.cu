
#include <hip/hip_runtime.h>
// dobra rešitev, neodvisna od tega števila blokov 
// število blokov lahko vnesemo kot argument ali pa jih izračunamo pred klicem ščepca

#ifdef __cplusplus
extern "C" {
#endif

__global__ void vectorSubtract4(float *c, const float *a, const float *b, int len) {
	// določimo globalni indeks elementov
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	// če je niti manj kot je dolžina vektorjev, morajo nekatere izračunati več razlik
	while (gid < len) {
		c[gid] = a[gid] - b[gid];
		gid += gridDim.x * blockDim.x;
	}
}

#ifdef __cplusplus
}
#endif
